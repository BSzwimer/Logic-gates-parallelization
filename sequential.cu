#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>


int operation(int a, int b, int opp) {
switch (opp)
{
case 0:
return a && b;
break;
case 1:
return a || b;
break;
case 2:
return !(a && b);
break;
case 3:
return !(a || b);
break;
case 4:
return a ^ b;
break;
case 5:
return !(a ^ b);
break;
}
}
int main(int argc, char* argv[])
{
char* file_name = argv[1];
char* file_solution = argv[3];
unsigned int file_length = atol(argv[2]);
//char* file_name = "C:\\Users\\bszwim\\Downloads\\input_10000.txt";
//char* file_solution = "C:\\Users\\bszwim\\Downloads\\my_solution.txt";
//unsigned int file_length = 10000;
FILE* file_answer = fopen(file_solution, "w");
FILE* file_one = fopen(file_name, "r");
if (file_one == NULL) {
perror("unable to open file");
exit(1);
}

char line[256];
int lineCounter = 0;
float memsettime;
hipEvent_t start, stop;
//clock_t t;

//t = clock();

hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start, 0);
hipEventSynchronize(start);
while (fgets(line, sizeof(line), file_one) && lineCounter < file_length) {
lineCounter++;
int a = line[0] - '0';
int b = line[2] - '0';
int opp = line[4] - '0';
int solution = operation(a, b, opp);
fprintf(file_answer, "%d\n", solution);


}
//t = clock() - t;
//double time_taken = ((double)t) / CLOCKS_PER_SEC; // in seconds 
//printf("%f seconds to execute \n", time_taken);
hipEventRecord(stop, 0);
hipEventSynchronize(stop);
hipEventElapsedTime(&memsettime, start, stop);
printf(" * Sequential execution time : %f * \n", memsettime);
hipEventDestroy(start);
hipEventDestroy(stop);


fclose(file_answer);
fclose(file_one);
return 0;
}