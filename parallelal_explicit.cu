#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <>
#include <string.h>
#include <ctype.h>
#define LENGTH 7
#define MAXSIZE 1000000

__global__ void operation(int* file_cuda, int* file_output_cuda, int arrSize) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int new_index = index * 3;
	//printf(" index: %d", index);
	if (index < (arrSize / 3)) {
		int a = file_cuda[new_index];
		int b = file_cuda[new_index + 1];
		int opp = file_cuda[new_index + 2];
		switch (opp)
		{
		case 0:
			file_output_cuda[index] = a && b;
			break;
		case 1:
			file_output_cuda[index] = a || b;
			break;
		case 2:
			file_output_cuda[index] = !(a && b);
			//return !(a && b);
			break;
		case 3:
			file_output_cuda[index] = !(a || b);
			//return !(a || b);
			break;
		case 4:
			file_output_cuda[index] = a ^ b;
			//return a ^ b;
			break;
		case 5:
			file_output_cuda[index] = !(a ^ b);
			//return !(a ^ b);
			break;
		}

	}
}
int main(int argc, char* argv[])
{
	float memsettime;
	hipEvent_t start, stop;
	char* file_name = argv[1];
	char* file_solution = argv[3];
	unsigned int file_length = atol(argv[2]);
	//char* file_name = "C:\\Users\\bszwim\\Downloads\\input_100000.txt";;
	//unsigned int file_length = 100000;
	//char* file_solution = "C:\\Users\\bszwim\\Downloads\\my_solution.txt";;
	FILE* file_answer = fopen(file_solution, "w");
	FILE* file_one = fopen(file_name, "r");
	if (file_one == NULL) {
		perror("unable to open file");
		exit(1);
	}

	int lines = 0;
	while (!feof(file_one))
	{
		char ch = fgetc(file_one);
		if (ch == '\n')
		{
			++lines;
		}
	}
	rewind(file_one);
	if (file_length < lines) {
		lines = file_length;
	}
	int* arr;
	arr = (int*)malloc((lines * 3) * sizeof(int));
	int counter = 0;
	int lineCounter = 0;
	while (!feof(file_one) && lineCounter < file_length) {
		char c = fgetc(file_one);
		if (c == '\n') {
			lineCounter++;
		}
		if (isdigit(c)) {
			int x = c - '0';
			arr[counter] = x;
			counter++;
		}
	}
	//arr[counter] = '\0';
	//printf("size of arr: %d \n", strlen(arr));
	//printf("%s", arr);
	int* file_cuda;
	int* file_output_cuda;
	hipMalloc((void**)&file_cuda, (lines * 3) * sizeof(int));
	hipMalloc((void**)&file_output_cuda, lines * sizeof(int));
	hipMemcpy(file_cuda, arr, (lines * 3) * sizeof(int), hipMemcpyHostToDevice);
	int* arr_output;
	arr_output = (int*)malloc(lines * sizeof(int));
	//int num_of_blocks = (lines / 1024) + 1;
	unsigned int num_of_blocks = (lines + 1024 - 1) / 1024;


	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	hipEventSynchronize(start);
	operation << < num_of_blocks, 1024 >> > (file_cuda, file_output_cuda, counter);


	
	hipMemcpy(arr_output, file_output_cuda, lines * sizeof(int), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&memsettime, start, stop);
	printf(" * CUDA execution time and data migration for explicit: %f * \n", memsettime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	for (int j = 0; j < lines; j++) {
		fprintf(file_answer, "%d\n", arr_output[j]);
	}


	hipFree(file_cuda);
	hipFree(file_output_cuda);
	free(arr_output);
	free(arr);


	fclose(file_answer);
	fclose(file_one);
	return 0;
}